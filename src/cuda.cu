#include "hip/hip_runtime.h"
#include "mymulmat.h"

#include <cstdio>
#include <hip/hip_runtime.h>

void printError(hipError_t e, int l) {
    if (e != hipSuccess) {
        printf("error: %s (code %d), line(%d)\n", hipGetErrorString(e), e, l);
        exit(EXIT_FAILURE);
    }
}

#define CheckError(f) printError((f), __LINE__)

__global__
void kernel(int n, int m, int k, melem_t *A, melem_t *B, melem_t *C) {
    for (int i = 0; i < n; i++) {
        for (int j= 0; j < m; j++) {
            for (int l = 0; l <k; l++) {
                C[i*m+j] += A[i*k+l] * B[l*m+j];
            }
        }
    }
}

uint64_t cudaGemm(int n, int m, int k, melem_t *A, melem_t *B, melem_t *C) {
    // device initialize
    int device = 0;
    hipSetDevice(device);

    // device malloc
    melem_t *devA, *devB, *devC;
    size_t sizeA = size_t(n)*k*sizeof(melem_t);
    size_t sizeB = size_t(k)*m*sizeof(melem_t);
    size_t sizeC = size_t(n)*m*sizeof(melem_t);

    CheckError(hipMalloc((void**) &devA, sizeA));
    CheckError(hipMalloc((void**) &devB, sizeB));
    CheckError(hipMalloc((void**) &devC, sizeC));

    // data load
    CheckError(hipMemcpy(devA, A, sizeA, hipMemcpyHostToDevice));
    CheckError(hipMemcpy(devB, B, sizeB, hipMemcpyHostToDevice));
    CheckError(hipMemcpy(devC, C, sizeC, hipMemcpyHostToDevice));

    // gemm start
    hipEvent_t start, stop;
    CheckError(hipEventCreate(&start));
    CheckError(hipEventCreate(&stop));
    hipDeviceSynchronize();

    // time measuring
    CheckError(hipEventRecord(start, NULL));
    kernel <<<1, 1>>> (n, m, k, devA, devB, devC);
    CheckError(hipEventRecord(stop, NULL));

    // gemm end
    CheckError(hipEventSynchronize(stop));
    float msec = 0.0f;
    CheckError(hipEventElapsedTime(&msec, start, stop));

    // data store
    CheckError(hipMemcpy(C, devC, sizeC, hipMemcpyDeviceToHost));

    // device free
    CheckError(hipFree(devA));
    CheckError(hipFree(devB));
    CheckError(hipFree(devC));

    return (uint64_t)(msec * 1000.0f);
}
