#include "hip/hip_runtime.h"
#include "mymulmat.h"

#include <cstdio>
#include <hip/hip_runtime.h>


void printError(hipError_t e, int l) {
    if (e != hipSuccess) {
        printf("error: %s (code %d), line(%d)\n", hipGetErrorString(e), e, l);
        exit(EXIT_FAILURE);
    }
}

#define CheckError(f) printError((f), __LINE__)

#define BX 32
#define BY 32
#define G_INDEX(b_x, b_y, t_x, t_y) (n * (BY * b_y + t_y) + (BX * b_x + t_x))
#define S_INDEX(t_x, t_y) (blockDim.x * t_y + t_x)

__global__
void kernel(int n, int m, int k, melem_t *A, melem_t *B, melem_t *C) {
    __shared__ melem_t A_[BX * BY];
    __shared__ melem_t B_[BX * BY];
    __shared__ melem_t C_[BX * BY];

    C_[S_INDEX(threadIdx.x, threadIdx.y)] = 0;
    for(int t = 0; t < n / BX; t++){
        A_[S_INDEX(threadIdx.x, threadIdx.y)] = A[G_INDEX(t, blockIdx.y, threadIdx.x, threadIdx.y)];
        B_[S_INDEX(threadIdx.x, threadIdx.y)] = B[G_INDEX(blockIdx.x, t, threadIdx.x, threadIdx.y)];
        __syncthreads();

        for(int s = 0; s < BX; s++){
            C_[S_INDEX(threadIdx.x, threadIdx.y)] += A_[S_INDEX(s, threadIdx.y)] * B_[S_INDEX(threadIdx.x, s)];
        }
        __syncthreads();
    }

    C[G_INDEX(blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y)] = C_[S_INDEX(threadIdx.x, threadIdx.y)];
}

uint64_t cudaGemm(int n, int m, int k, melem_t *A, melem_t *B, melem_t *C) {
    // device initialize
    int device = 0;
    hipSetDevice(device);

    // device malloc
    melem_t *devA, *devB, *devC;
    size_t sizeA = size_t(n)*k*sizeof(melem_t);
    size_t sizeB = size_t(k)*m*sizeof(melem_t);
    size_t sizeC = size_t(n)*m*sizeof(melem_t);

    CheckError(hipMalloc((void**) &devA, sizeA));
    CheckError(hipMalloc((void**) &devB, sizeB));
    CheckError(hipMalloc((void**) &devC, sizeC));

    // data load
    CheckError(hipMemcpy(devA, A, sizeA, hipMemcpyHostToDevice));
    CheckError(hipMemcpy(devB, B, sizeB, hipMemcpyHostToDevice));
    CheckError(hipMemcpy(devC, C, sizeC, hipMemcpyHostToDevice));

    // gemm start
    hipEvent_t start, stop;
    CheckError(hipEventCreate(&start));
    CheckError(hipEventCreate(&stop));
    hipDeviceSynchronize();

    dim3 grid(n / BX, n / BY);
    dim3 block(BX, BY);

    // time measuring
    CheckError(hipEventRecord(start, NULL));
    kernel <<<grid, block>>> (n, m, k, devA, devB, devC);
    CheckError(hipEventRecord(stop, NULL));

    // gemm end
    CheckError(hipEventSynchronize(stop));
    float msec = 0.0f;
    CheckError(hipEventElapsedTime(&msec, start, stop));

    // data store
    CheckError(hipMemcpy(C, devC, sizeC, hipMemcpyDeviceToHost));

    // device free
    CheckError(hipFree(devA));
    CheckError(hipFree(devB));
    CheckError(hipFree(devC));

    return (uint64_t)(msec * 1000.0f);
}
