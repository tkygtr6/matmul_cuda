#include "hip/hip_runtime.h"
#include "mymulmat.h"

#include <cstdio>
#include <hip/hip_runtime.h>


void printError(hipError_t e, int l) {
    if (e != hipSuccess) {
        printf("error: %s (code %d), line(%d)\n", hipGetErrorString(e), e, l);
        exit(EXIT_FAILURE);
    }
}

#define CheckError(f) printError((f), __LINE__)

#define BX 32
#define BY 32
#define G_INDEX(b_x, b_y, t_x, t_y) ((n) * ((BY) * (b_y) + (t_y)) + ((BX) * (b_x) + (t_x)))
#define S_INDEX_1(t_x, t_y) ((BX) * (t_y) + (t_x))
#define S_INDEX_2(t_x, t_y) ((BX) * 2 * (t_y) + (t_x))

__global__
void kernel(int n, int m, int k, melem_t *A, melem_t *B, melem_t *C) {
    __shared__ melem_t A_[BX * BY];
    __shared__ melem_t B_[(BX * 2) * BY];
    __shared__ melem_t C_[(BX * 2) * BY];

    C_[S_INDEX_2(threadIdx.x, threadIdx.y)] = 0;
    C_[S_INDEX_2(threadIdx.x + BX, threadIdx.y)] = 0;
    for(int t = 0; t < n / BX; t++){
        A_[S_INDEX_1(threadIdx.x, threadIdx.y)] = A[G_INDEX(t, blockIdx.y, threadIdx.x, threadIdx.y)];
        B_[S_INDEX_2(threadIdx.x, threadIdx.y)] = B[G_INDEX(blockIdx.x * 2, t, threadIdx.x, threadIdx.y)];
        B_[S_INDEX_2(threadIdx.x + BX, threadIdx.y)] = B[G_INDEX(blockIdx.x * 2 + 1, t, threadIdx.x, threadIdx.y)];
        __syncthreads();

        for(int s = 0; s < BX; s++){
            C_[S_INDEX_2(threadIdx.x, threadIdx.y)] += A_[S_INDEX_1(s, threadIdx.y)] * B_[S_INDEX_2(threadIdx.x, s)];
            C_[S_INDEX_2(threadIdx.x + BX, threadIdx.y)] += A_[S_INDEX_1(s, threadIdx.y)] * B_[S_INDEX_2(threadIdx.x + BX, s)];
        }
        __syncthreads();
    }

    C[G_INDEX(blockIdx.x * 2, blockIdx.y, threadIdx.x, threadIdx.y)] = C_[S_INDEX_2(threadIdx.x, threadIdx.y)];
    C[G_INDEX(blockIdx.x * 2 + 1, blockIdx.y, threadIdx.x, threadIdx.y)] = C_[S_INDEX_2(threadIdx.x + BX, threadIdx.y)];
}

uint64_t cudaGemm(int n, int m, int k, melem_t *A, melem_t *B, melem_t *C) {
    // device initialize
    int device = 0;
    hipSetDevice(device);

    // device malloc
    melem_t *devA, *devB, *devC;
    size_t sizeA = size_t(n)*k*sizeof(melem_t);
    size_t sizeB = size_t(k)*m*sizeof(melem_t);
    size_t sizeC = size_t(n)*m*sizeof(melem_t);

    CheckError(hipMalloc((void**) &devA, sizeA));
    CheckError(hipMalloc((void**) &devB, sizeB));
    CheckError(hipMalloc((void**) &devC, sizeC));

    // data load
    CheckError(hipMemcpy(devA, A, sizeA, hipMemcpyHostToDevice));
    CheckError(hipMemcpy(devB, B, sizeB, hipMemcpyHostToDevice));
    CheckError(hipMemcpy(devC, C, sizeC, hipMemcpyHostToDevice));

    // gemm start
    hipEvent_t start, stop;
    CheckError(hipEventCreate(&start));
    CheckError(hipEventCreate(&stop));
    hipDeviceSynchronize();

    dim3 grid(n / (BX * 2), n / BY);
    dim3 block(BX, BY);

    // time measuring
    CheckError(hipEventRecord(start, NULL));
    kernel <<<grid, block>>> (n, m, k, devA, devB, devC);
    CheckError(hipEventRecord(stop, NULL));

    // gemm end
    CheckError(hipEventSynchronize(stop));
    float msec = 0.0f;
    CheckError(hipEventElapsedTime(&msec, start, stop));

    // data store
    CheckError(hipMemcpy(C, devC, sizeC, hipMemcpyDeviceToHost));

    // device free
    CheckError(hipFree(devA));
    CheckError(hipFree(devB));
    CheckError(hipFree(devC));

    return (uint64_t)(msec * 1000.0f);
}
