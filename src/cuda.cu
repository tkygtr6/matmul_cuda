#include "hip/hip_runtime.h"
#include "mymulmat.h"

#include <cstdio>
#include <hip/hip_runtime.h>


void printError(hipError_t e, int l) {
    if (e != hipSuccess) {
        printf("error: %s (code %d), line(%d)\n", hipGetErrorString(e), e, l);
        exit(EXIT_FAILURE);
    }
}

#define CheckError(f) printError((f), __LINE__)

#define BX 32
#define BY 32
#define UNX 1
#define UNY 1
#define G_INDEX(b_x, b_y, t_x, t_y) ((n) * ((BY) * (b_y) + (t_y)) + ((BX) * (b_x) + (t_x)))
#define S_INDEX(t_x, t_y, w_x) ((BX) * (t_y) * (w_x) + (t_x))

__global__
void kernel(int n, int m, int k, melem_t *A, melem_t *B, melem_t *C) {
    __shared__ melem_t A_[BX * (BY * UNY)];
    __shared__ melem_t B_[(BX * UNX) * BY];
    __shared__ melem_t C_[(BX * UNX) * (BY * UNY)];

    for(int i = 0; i < UNX; i++){
        for(int j = 0; j < UNY; j++){
            C_[S_INDEX(threadIdx.x + BX * i, threadIdx.y + BY * j, UNX)] = 0;
        }
    }
    for(int t = 0; t < n / BX; t++){
        for(int i = 0; i < UNY; i++){
            A_[S_INDEX(threadIdx.x, threadIdx.y + BY * i, 1)] = A[G_INDEX(t, blockIdx.y * UNY + i, threadIdx.x, threadIdx.y)];
        }
        for(int i = 0; i < UNX; i++){
            B_[S_INDEX(threadIdx.x + BX * i, threadIdx.y, UNX)] = B[G_INDEX(blockIdx.x * UNX + i, t, threadIdx.x, threadIdx.y)];
        }
        __syncthreads();

        for(int s = 0; s < BX; s++){
            for(int i = 0; i < UNX; i++){
                for(int j = 0; j < UNY; j++){
                    C_[S_INDEX(threadIdx.x + BX * i, threadIdx.y + BY * j, UNX)] += A_[S_INDEX(s, threadIdx.y + BY * j, 1)] * B_[S_INDEX(threadIdx.x + BX * i, s, UNX)];
                }
            }
        }
        __syncthreads();
    }

    for(int i = 0; i < UNX; i++){
        for(int j = 0; j < UNY; j++){
            C[G_INDEX(blockIdx.x * UNX + i, blockIdx.y * UNY + j, threadIdx.x, threadIdx.y)] = C_[S_INDEX(threadIdx.x + BX * i, threadIdx.y + BY * j, UNX)];
        }
    }
}

uint64_t cudaGemm(int n, int m, int k, melem_t *A, melem_t *B, melem_t *C) {
    // device initialize
    int device = 0;
    hipSetDevice(device);

    // device malloc
    melem_t *devA, *devB, *devC;
    size_t sizeA = size_t(n)*k*sizeof(melem_t);
    size_t sizeB = size_t(k)*m*sizeof(melem_t);
    size_t sizeC = size_t(n)*m*sizeof(melem_t);

    CheckError(hipMalloc((void**) &devA, sizeA));
    CheckError(hipMalloc((void**) &devB, sizeB));
    CheckError(hipMalloc((void**) &devC, sizeC));

    // data load
    CheckError(hipMemcpy(devA, A, sizeA, hipMemcpyHostToDevice));
    CheckError(hipMemcpy(devB, B, sizeB, hipMemcpyHostToDevice));
    CheckError(hipMemcpy(devC, C, sizeC, hipMemcpyHostToDevice));

    // gemm start
    hipEvent_t start, stop;
    CheckError(hipEventCreate(&start));
    CheckError(hipEventCreate(&stop));
    hipDeviceSynchronize();

    dim3 grid(n / (BX * UNX), n / (BY * UNY));
    dim3 block(BX, BY);

    // time measuring
    CheckError(hipEventRecord(start, NULL));
    kernel <<<grid, block>>> (n, m, k, devA, devB, devC);
    CheckError(hipEventRecord(stop, NULL));

    // gemm end
    CheckError(hipEventSynchronize(stop));
    float msec = 0.0f;
    CheckError(hipEventElapsedTime(&msec, start, stop));

    // data store
    CheckError(hipMemcpy(C, devC, sizeC, hipMemcpyDeviceToHost));

    // device free
    CheckError(hipFree(devA));
    CheckError(hipFree(devB));
    CheckError(hipFree(devC));

    return (uint64_t)(msec * 1000.0f);
}
