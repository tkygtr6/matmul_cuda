#include "hip/hip_runtime.h"
#include "mymulmat.h"

#include <cstdio>
#include <hip/hip_runtime.h>


void printError(hipError_t e, int l) {
    if (e != hipSuccess) {
        printf("error: %s (code %d), line(%d)\n", hipGetErrorString(e), e, l);
        exit(EXIT_FAILURE);
    }
}

#define CheckError(f) printError((f), __LINE__)

#define BX 32
#define BY 8
#define STEP 32
#define UNX 8
#define UNY 16
#define G_A_INDEX(b_x, b_y, t_x, t_y) ((n) * ((BY) * (b_y) + (t_y)) + ((STEP) * (b_x) + (t_x)))
#define G_B_INDEX(b_x, b_y, t_x, t_y) ((n) * ((STEP) * (b_y) + (t_y)) + ((BX) * (b_x) + (t_x)))
#define G_C_INDEX(b_x, b_y, t_x, t_y) ((n) * ((BY) * (b_y) + (t_y)) + ((BX) * (b_x) + (t_x)))
#define S_INDEX(t_x, t_y, w_x) ((t_y) * (w_x) + (t_x))

__global__
void kernel(int n, int m, int k, melem_t *A, melem_t *B, melem_t *C) {
    __shared__ melem_t A_[STEP * (BY * UNY)];
    __shared__ melem_t B_[(BX * UNX) * STEP];
    melem_t C_[UNX * UNY];
    for(int i = 0; i < UNX; i++){
        for(int j = 0; j < UNY; j++){
            C_[S_INDEX(i, j, UNX)] = 0;
        }
    }
    for(int s = 0; s < n / STEP; s++){
        for(int t = 0; t < STEP/BX; t++){
            for(int i = 0; i < UNY; i++){
                A_[S_INDEX(threadIdx.x + BX * t, threadIdx.y + BY * i, STEP)] = A[G_A_INDEX(s, blockIdx.y * UNY + i, threadIdx.x + BX * t, threadIdx.y)];
            }
        }
        for(int t = 0; t < STEP/BY; t++){
            for(int i = 0; i < UNX; i++){
                B_[S_INDEX(threadIdx.x + BX * i, threadIdx.y + BY * t, BX * UNX)] = B[G_B_INDEX(blockIdx.x * UNX + i, s, threadIdx.x, threadIdx.y + BY * t)];
            }
        }
        __syncthreads();

        for(int k = 0; k < STEP; k++){
            for(int i = 0; i < UNX; i++){
                for(int j = 0; j < UNY; j++){
                    C_[S_INDEX(i, j, UNX)] += A_[S_INDEX(k, threadIdx.y + BY * j, STEP)] * B_[S_INDEX(threadIdx.x + BX * i, k, BX * UNX)];
                }
            }
        }
        __syncthreads();
    }

    for(int i = 0; i < UNX; i++){
        for(int j = 0; j < UNY; j++){
            C[G_C_INDEX(blockIdx.x * UNX + i, blockIdx.y * UNY + j, threadIdx.x, threadIdx.y)] = C_[S_INDEX(i, j, UNX)];
        }
    }
}

uint64_t cudaGemm(int n, int m, int k, melem_t *A, melem_t *B, melem_t *C) {
    // device initialize
    int device = 0;
    hipSetDevice(device);

    // device malloc
    melem_t *devA, *devB, *devC;
    size_t sizeA = size_t(n)*k*sizeof(melem_t);
    size_t sizeB = size_t(k)*m*sizeof(melem_t);
    size_t sizeC = size_t(n)*m*sizeof(melem_t);

    CheckError(hipMalloc((void**) &devA, sizeA));
    CheckError(hipMalloc((void**) &devB, sizeB));
    CheckError(hipMalloc((void**) &devC, sizeC));

    // data load
    CheckError(hipMemcpy(devA, A, sizeA, hipMemcpyHostToDevice));
    CheckError(hipMemcpy(devB, B, sizeB, hipMemcpyHostToDevice));
    CheckError(hipMemcpy(devC, C, sizeC, hipMemcpyHostToDevice));

    // gemm start
    hipEvent_t start, stop;
    CheckError(hipEventCreate(&start));
    CheckError(hipEventCreate(&stop));
    hipDeviceSynchronize();

    dim3 grid(n / (BX * UNX), n / (BY * UNY));
    dim3 block(BX, BY);
    printf("Shared memory: %d\n", (STEP * BY * UNY + BX * UNX * STEP) * 4);

    // time measuring
    CheckError(hipEventRecord(start, NULL));
    kernel <<<grid, block>>> (n, m, k, devA, devB, devC);
    CheckError(hipEventRecord(stop, NULL));

    // gemm end
    CheckError(hipEventSynchronize(stop));
    float msec = 0.0f;
    CheckError(hipEventElapsedTime(&msec, start, stop));

    // data store
    CheckError(hipMemcpy(C, devC, sizeC, hipMemcpyDeviceToHost));

    // device free
    CheckError(hipFree(devA));
    CheckError(hipFree(devB));
    CheckError(hipFree(devC));

    return (uint64_t)(msec * 1000.0f);
}
